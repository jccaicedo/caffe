#include "hip/hip_runtime.h"
// Copyright 2014 Juan C. Caicedo

#include <cmath>
#include <cstdlib>
#include <cstring>

#include <stdio.h>

#include "caffe/common.hpp"
#include "caffe/util/regions.hpp"

namespace caffe {

// Kernel to copy an image already in the GPU to the Blob
template <typename Dtype>
__global__ void copyRegionToBlob_kernel(const unsigned char* sourceData, 
                                        Dtype* destData, size_t srcstep, 
                                        int region, int rows, int cols,
                                        int channels, int cropsize, const Dtype* meanImg,
					bbox padding) {
  const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int w_off = 14, h_off = 14, meanSize = 256;

  if((padding.x1 <= xIndex) && (padding.y1 <= yIndex) &&
     (xIndex < cols - padding.x2) && (yIndex < rows - padding.y2)){
    const int pixelId = yIndex * srcstep + (3 * xIndex);
    for(int c = 0; c < channels; ++c) {
      int meanImgCoord = (c * meanSize + yIndex + h_off) * meanSize + xIndex + w_off;
      destData[((region * channels + c) * cropsize + yIndex) * cropsize + xIndex] 
          = static_cast<Dtype>(sourceData[pixelId + c])
            - meanImg[ meanImgCoord ];

      //if((xIndex < 100 && xIndex > 95) && (yIndex < 100 && yIndex > 95))
      //if((xIndex == 0) && (yIndex == 0))
      //printf("pixelCoord %d,%d => %d meanImgCoord => %d\n", yIndex,xIndex, pixelId+c ,meanImgCoord);
      //printf("meanPixelCoord %d,%d,%d => %d meanPixelValue => %f\n", yIndex,xIndex,c, meanImgCoord, meanImg[ meanImgCoord ]);
    }
  }
}

// Kernel to copy a portion of the blob to a GPU image
template <typename Dtype>
__global__ void copyBlobToRegion_kernel(const Dtype* blob, unsigned char* image,
                                        size_t srcstep, int region, int rows, int cols, 
                                        int channels, int cropsize, const Dtype* meanImg,
					bbox padding) {
  const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  int w_off = 14, h_off = 14, meanSize = 256;

   if( (padding.x1 <= xIndex) && (padding.y1 <= yIndex) && 
       (xIndex < cols - padding.x2) && (yIndex < rows - padding.y2)){
    const int pixelId = yIndex * srcstep + (3 * xIndex);
    for(int c = 0; c < channels; ++c) {
      int meanImgCoord = (c * meanSize + yIndex + h_off) * meanSize + xIndex + w_off;
      image[pixelId + c] = static_cast<unsigned char>(
      blob[((region * channels + c) * cropsize + yIndex) * cropsize + xIndex]
      + meanImg[ meanImgCoord ] );
    }
  }
}

// Call to the kernel to copy from region to blob
template <typename Dtype>
void copyRegionToBlob(const unsigned char* sourceData, Dtype* destData,
                      size_t srcstep, int region, int rows, int cols, 
                      int channels, int cropsize, const Dtype* meanImg, bbox padding) {
  dim3 blockD(32, 32);
  const dim3 grid((cols + blockD.x - 1)/blockD.x, (rows + blockD.y - 1)/blockD.y);
  copyRegionToBlob_kernel<<<grid, blockD>>>(
                                sourceData, destData, srcstep, region, 
                                rows, cols, channels, cropsize, meanImg, padding);
  CUDA_POST_KERNEL_CHECK;
}

// Call to the kernel to copy from blob to region
template <typename Dtype>
void copyBlobToRegion(const Dtype* blob, unsigned char* image,
                      size_t srcstep, int region, int rows, int cols, 
                      int channels, int cropsize, const Dtype* meanImg, bbox padding) {
  dim3 blockD(32, 32);
  const dim3 grid((cols + blockD.x - 1)/blockD.x, (rows + blockD.y - 1)/blockD.y);
  copyBlobToRegion_kernel<<<grid, blockD>>>(
                                  blob, image, srcstep, region, rows,
                                  cols, channels, cropsize, meanImg, padding);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiations
template 
__global__ void copyRegionToBlob_kernel<float>(const unsigned char* sourceData,
                                        float* destData, size_t srcstep, 
                                        int region, int rows, int cols,
                                        int channels, int cropsize, const float* meanImg,
					bbox padding);
template
void copyRegionToBlob<float>(const unsigned char* sourceData, float* destData,
                      size_t srcstep, int region, int rows, int cols,
                      int channels, int cropsize, const float* meanImg, bbox padding);


template
__global__ void copyBlobToRegion_kernel<float>(const float* blob, unsigned char* image,
                                        size_t srcstep, int region, int rows,
                                        int cols, int channels, int cropsize, const float* meanImg,
					bbox padding);

template
void copyBlobToRegion<float>(const float* blob, unsigned char* image,
                             size_t srcstep, int region, int rows, int cols, 
                             int channels, int cropsize, const float* meanImg, bbox padding);


} // namespace caffe
