#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
//#include "caffe/vision_layers.hpp"
#include "caffe/q_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include <cmath>

using std::max;
using std::abs;

namespace caffe {

template <typename Dtype>
__global__ void kernel_max_div(const int num, const int dim,
    const Dtype* scale, Dtype* data) {
  CUDA_KERNEL_LOOP(index, num * dim) {
    int n = index / dim;
    data[index] /= scale[n];
  }
}

template <typename Dtype>
__global__ void kernel_get_absmax(const int num, const int dim, 
    const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num) {
    Dtype absMax = -FLT_MAX;
    for (int i = 0; i < dim; ++i) {
      absMax = max( abs(data[index * dim + i]), absMax);// * data[index * dim + i];
    }
    out[index] = absMax;
  }
}

template <typename Dtype>
void QLearningLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int num = bottom[0]->num();
  int dim = bottom[0]->count() / bottom[0]->num();
  CUDA_CHECK(hipMemcpy(top_data, bottom_data,
      sizeof(Dtype) * bottom[0]->count(), hipMemcpyDeviceToDevice));
  // We need to normalize output vector to avoid numerical issues
  // Compute dot products
  kernel_get_absmax<Dtype><<<CAFFE_GET_BLOCKS(num), CAFFE_CUDA_NUM_THREADS>>>(
    num, dim, bottom_data, scale_data);
  // Do division
  kernel_max_div<Dtype><<<CAFFE_GET_BLOCKS(num * dim),
                              CAFFE_CUDA_NUM_THREADS>>>(
      num, dim, scale_data, top_data);
}

// TODO(Yangqing): implement the GPU version of softmax.
template <typename Dtype>
Dtype QLearningLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  int num = top[0]->num();
  int dim = top[0]->count() / top[0]->num();
  CUDA_CHECK(hipMemcpy(bottom_diff, top_diff,
      sizeof(Dtype) * top[0]->count(), hipMemcpyDeviceToDevice));
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff
  // cuda dot returns the result to cpu, so we temporarily change the pointer
  // mode
  /*CUBLAS_CHECK(hipblasSetPointerMode(Caffe::cublas_handle(),
      HIPBLAS_POINTER_MODE_DEVICE));
  Dtype* scale_data = scale_.mutable_gpu_data();
  for (int i = 0; i < num; ++i) {
    caffe_gpu_dot<Dtype>(dim, top_diff + i * dim,
        top_data + i * dim, scale_data + i);
  }
  CUBLAS_CHECK(hipblasSetPointerMode(Caffe::cublas_handle(),
      HIPBLAS_POINTER_MODE_HOST));
  // subtraction
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, -1.,
      scale_.gpu_data(), sum_multiplier_.gpu_data(), 1., bottom_diff);*/
  // elementwise multiplication

  LOG(INFO) << "Backward GPU not implemented for QLearningLayer";
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
  return Dtype(0);
}

INSTANTIATE_CLASS(QLearningLayer);


}  // namespace caffe
